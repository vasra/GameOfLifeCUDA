#include "hip/hip_runtime.h"
#include <gol.cuh>
#include <assert.h>
#include <iostream>
#include <vector>
#include <array>
#include <string>
#include <cmath>

void copyHaloRowsTest();
void copyHaloColumnsTest();
bool compareFirstRealAndBottomHaloRow(char* h_life, int size);
bool compareLastRealAndTopHaloRow(char* h_life, int size);

// the size of the grid without the halos
constexpr double size = 25.0f;
constexpr double threads = 16.0f;

// The four corners of the grid that contain REAL elements and not halo elements
constexpr int topLeft     = size + 3;
constexpr int topRight    = topLeft + size - 1;
constexpr int bottomLeft  = (size + 2) * size + 1;
constexpr int bottomRight = bottomLeft + size - 1;

constexpr int bottomLeftHalo  = (size + 1) * (size + 2) + 1;
constexpr int bottomRightHalo = bottomLeftHalo + size - 1;

int
main() {
    std::cout << "Hello world test!" << std::endl;
    std::cout << "Top left element index     : " << topLeft << std::endl;
    std::cout << "Top right element index    : " << topRight << std::endl;
    std::cout << "Bottom left element index  : " << bottomLeft << std::endl;
    std::cout << "Bottom right element index : " << bottomRight << std::endl;
    copyHaloRowsTest();
    //copyHaloColumnsTest();
    return 0;
}

void
copyHaloRowsTest() {
  char* h_life = (char*)malloc((size + 2) * (size + 2) * sizeof(char));
  assert(h_life != NULL);
  initialState(size, h_life);
  //printGrid(size, h_life);

  char* d_life;
  hipError_t err;
  err = hipMalloc((void**)&d_life, (size + 2) * (size + 2) * sizeof(char));
  assert(hipSuccess == err);

  err = hipMemcpy(d_life, h_life, (size + 2) * (size + 2) * sizeof(char), hipMemcpyHostToDevice);
  assert(hipSuccess == err);

  int copyingBlocksRows = static_cast<int>(ceil(size / threads));
  std::cout << "copyingBlocksRows " << copyingBlocksRows << std::endl;
  bool ret = compareLastRealAndTopHaloRow(h_life, size);
  ret = compareFirstRealAndBottomHaloRow(h_life, size);

  copyHaloRows<<<copyingBlocksRows, threads>>>(d_life, size);

  err = hipDeviceSynchronize();
  assert(hipSuccess == err);

  err = hipMemcpy(h_life, d_life, (size + 2) * (size + 2) * sizeof(char), hipMemcpyDeviceToHost);
  assert(hipSuccess == err);
  printGrid(size, h_life);

  hipFree(d_life);
  free(h_life);
}

// void copyHaloColumnsTest() {
//     char* h_life = (char*)malloc((size + 2) * (size + 2) * sizeof(char));
//     assert(h_life != NULL);
//     initialState(size, h_life);

//     char* d_life;
//     hipError_t err;
//     err = hipMalloc((void**)&d_life, (size + 2) * (size + 2) * sizeof(char));
//     assert(hipSuccess == err);

//     err = hipMemcpy(d_life, h_life, (size + 2) * (size + 2) * sizeof(char), hipMemcpyHostToDevice);
//     assert(hipSuccess == err);

//     constexpr int copyingBlocksColumns = size / threads;
//     std::vector<char> firstRealColumn;
//     std::vector<char> lastRealColumn;

//     // copy bottom-right corner element
//     firstRealColumn.push_back(*(h_life + size * (size + 2) + size));

//     // copy bottom-left corner element
//     lastRealColumn.push_back(*(h_life + size * (size + 2) + 1));

//     // copy rest of the elements
//     for (int i = 1; i < size + 1; i++) {
//         firstRealColumn.push_back(*(h_life + i * (size + 2) + 1));
//         lastRealColumn.push_back(*(h_life + i * (size + 2) + size));
//     }

//     // copy top-right corner element
//     firstRealColumn.push_back(*(h_life + size * 2));

//     // copy top-left corner element
//     lastRealColumn.push_back(*(h_life + size + 3));

//     assert(firstRealColumn.size() == size + 2);
//     assert(lastRealColumn.size() == size + 2);

//     copyHaloRows<<<copyingBlocksColumns, threads>>>(d_life, size);

//     err = hipDeviceSynchronize();
//     assert(hipSuccess == err);

//     err = hipMemcpy(h_life, d_life, (size + 2) * (size + 2) * sizeof(char), hipMemcpyDeviceToHost);
//     assert(hipSuccess == err);

//     std::vector<char> leftHaloColumn;
//     std::vector<char> rightHaloColumn;

//     // copy halo columns
//     for (int i = 0; i < size + 2; i++) {
//         leftHaloColumn.push_back(*(h_life + i * (size + 2)));
//         rightHaloColumn.push_back(*(h_life + i * (size + 2) + size + 1));
//     }

//     assert(size + 2 == leftHaloColumn.size());
//     assert(size + 2 == rightHaloColumn.size());

//     assert(firstRealColumn == rightHaloColumn);
//     assert(lastRealColumn == leftHaloColumn);

//     printGrid(size, h_life);
//     hipFree(d_life);
//     free(h_life);
// }

bool
compareFirstRealAndBottomHaloRow(char* h_life, int size) {
  // Indices of the first element in the first REAL row, and the
  // first element in the bottom halo row respectively. We do not take
  // into account the corner elements
  int topLeftReal = size + 3;
  int bottomLeftHalo = (size + 1) * (size + 2) + 1;

  for (int i = 0; i < size; i++) {
    if (*(h_life + i + size + 2) != *(h_life + size * (size + 2) + i)) {
      return false;
    }
  }
  return true;
}

bool
compareLastRealAndTopHaloRow(char* h_life, int size) {
  // Indices of the first element in the last REAL row, and the
  // first element in the top halo row respectively. We do not take
  // into account the corner elements
  int bottomLeftReal = size * (size + 2) + 1;
  int topLeftHalo = 1;

  for (int i = 0; i < size; i++) {
    if (*(h_life + bottomLeftReal + i) != *(h_life + topLeftHalo + i)) {
      return false;
    }
  }
  return true;
}
